#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vecSquareKernel(float *input, float *output, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        output[idx] = input[idx] * input[idx];
    }
}

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b)
{
    return (a + b - 1) / b;
}

// host memory manage
__global__ void vecSquare_1(float *input, float *output, int n)
{
    float *a_d, *b_d;
    size_t size = n * sizeof(float);

    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);

    hipMemcpy(a_d, input, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, output, size, hipMemcpyHostToDevice);

    const unsigned int threadNum = 256;
    unsigned int blockNum = cdiv(n, threadNum);

    vecSquareKernel<<<blockNum, threadNum>>>(a_d, b_d, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipMemcpy(output, b_d, size, hipMemcpyDeviceToHost);

    hipFree(a_d);
    hipFree(b_d);
}

// unified memory manage and data prefetch
__global__ void vecSquare_2(float *input, float *output, int n)
{
    int deviceId;
    hipGetDevice(&deviceId);
    size_t size = n * sizeof(float);

    hipMallocManaged(&input, size);
    hipMallocManaged(&output, size);

    hipMemPrefetchAsync(input, size, deviceId);

    const unsigned int threadNum = 256;
    unsigned int blockNum = cdiv(n, threadNum);

    vecSquareKernel<<<blockNum, threadNum>>>(input, output, n);
    hipMemPrefetchAsync(output, size, hipCpuDeviceId);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipFree(input);
    hipFree(output);
}